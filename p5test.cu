
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <math.h>
#include <ctime>
#include <vector>
using namespace std;

#define N 2000
#define count N*N
#define threadsPerBlock 1000
#define numberBlocks N*N/threadsPerBlock

// __device__ int partition(double* input, int start, int end)
// {
//     double pivot = input[end];
//
//     while(start < end){
//         while(input[start] < pivot)
//             start++;
//         while (input[end] > pivot)
//             end--;
//         if (input[start] == inpucout<<"time: "<<endTime<<"   "<<startTime<<"   "<<CLOCKS_PER_SEC<<endl;t[end])
//             start++;
//         else if(start < end){
//             double tmp = input[start];
//             input[start] = input[end];
//             input[end] = tmp;
//         }
//     }
//     return end;
// }
//
// __device__ double quickSelect(double* input, int p, int r, int k){
//     if(p == r){
//       return input[p];
//     }
//     int j = partition(input, p, r);
//     int length = j - p + 1;
//     if (length == k){
//       return input[j];
//     }cout<<"time: "<<endTime<<"   "<<startTime<<"   "<<CLOCKS_PER_SEC<<endl;
//     else if( k < length ){
//       return quickSelect(input, p, j - 1, k);
//     }
//     else{
//       return quickSelect(input, j + 1, r, k - length);
//     }
// }

__device__ void sort(double* input){
  for(int i=0;i<5;i++){
    for(int j=i;j<5;j++){
      if(input[j]<input[i]){
        double new_temp = input[i];
        input[i] = input[j];
        input[j] = new_temp;
      }
    }
  }
}

__global__ void median (double *a, double *b) {
  int number = blockIdx.x*blockDim.x + threadIdx.x;
  // if((number <N) || (number>=N*N-N)||(number/N==0)||(number/N==N-1)){
  //   b[number]=a[number];
  // }
  // if((number > N-1) && (threadIdx.x > 0) && (threadIdx.x < N-1) && (number < N*N-N)){
  if((number > N-1) && (number%N > 0) && (number%N < N-1) && (number < N*N-N)){
    double tempCompare[5];
    tempCompare[0] = a[number];
    tempCompare[1] = a[number-1];
    tempCompare[2] = a[number+1];
    tempCompare[3] = a[number-N];
    tempCompare[4] = a[number+N];
    // b[number] = quickSelect(tempCompare,0,4,2);
    // a[number] = tempCompare[2];
    sort(tempCompare);
    b[number]=tempCompare[2];
  }
  else if(number < N*N){
    b[number]=a[number];
  }
  __syncthreads();
}

__global__ void move (double *b, double *a) {
  int number = blockIdx.x*blockDim.x + threadIdx.x;
  a[number] = b[number];
}

__global__ void reduction (double *in, double *out) {
  __shared__ double temp[threadsPerBlock];
  int id = threadIdx.x;
  temp[id] = in[blockIdx.x*blockDim.x + id];
  __syncthreads();
  if(id<500 && id>11){
    temp[id] += temp[id+500]; __syncthreads();
  }
  __syncthreads();
  if(id<256){
    temp[id] += temp[id+256]; __syncthreads();
  }
  if(id<128){
    temp[id] += temp[id+128]; __syncthreads();
  }
  if(id<64){
    temp[id] += temp[id+64]; __syncthreads();
  }
  if(id<32){
    temp[id] += temp[id+32]; __syncthreads();
  }
  if(id<16){
    temp[id] += temp[id+16]; __syncthreads();
  }
  if(id<8){
    temp[id] += temp[id+8]; __syncthreads();
  }
  if(id<4){
    temp[id] += temp[id+4]; __syncthreads();
  }
  if(id<2){
    temp[id] += temp[id+2]; __syncthreads();
  }
  if(id<1){
    temp[id] += temp[id+1]; __syncthreads();
  }
  if(id<1){out[blockIdx.x] = temp[id];}
}

__global__ void sumGen (double *in, double *out) {
  for(int i=0;i<(N/threadsPerBlock)*(N/threadsPerBlock);i++){
    out[0]+=in[i];
  }
}

__global__ void assign (double *a, double *spe) {
  spe[0] = a[count/2+N/2];
  spe[1] = a[17*N+31];
}

int main(){
  double A[count], B[count];
  double sum[1], speNum[2];
  double *d_a, *d_b, *d_partSum, *d_ppartSum, *d_sum, *d_speNum;
  int size = N*N*sizeof(double);
  int twosize = 2*sizeof(double);

  sum[0]=0;

  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      A[i*N+j] = sin(i*i+j)*sin(i*i+j)+cos(i-j);
      // A[i*N+j] = j;
      B[i*N+j] = 0;
    }
  }
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_partSum, size/threadsPerBlock);
  hipMalloc((void **)&d_ppartSum, size/threadsPerBlock/threadsPerBlock);
  hipMalloc((void **)&d_sum, sizeof(double));
  hipMalloc((void **)&d_speNum,twosize);
  hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_sum, sum, sizeof(double), hipMemcpyHostToDevice);
  // clock_t startaaa = clock();
  hipEvent_t startTime=0, endTime=0;
  hipEventCreate(&startTime);
  hipEventCreate(&endTime);
  // auto start = std::chrono::system_clock::now();
  hipEventRecord(startTime, 0);
  for(int i=0;i<10;i++){
      median<<<numberBlocks,threadsPerBlock>>>(d_a,d_b);
      hipDeviceSynchronize();
      move<<<numberBlocks,threadsPerBlock>>>(d_b,d_a);
      hipDeviceSynchronize();
  }
  reduction<<<count/threadsPerBlock, threadsPerBlock>>>(d_a,d_partSum);
  reduction<<<(count/threadsPerBlock/threadsPerBlock),threadsPerBlock>>>(d_partSum,d_ppartSum);
  sumGen<<<1,1>>>(d_ppartSum,d_sum);
  assign<<<1,1>>>(d_a, d_speNum);
  hipDeviceSynchronize();
  // clock_t endbbb = clock();
  hipEventRecord(endTime, 0);
  hipEventSynchronize(endTime) ;
  float time;
  hipEventElapsedTime(&time,startTime,endTime);
  // auto end = std::chrono::system_clock::now();
  // std::chrono::duration<double> elapsed_seconds = end-start;

  hipMemcpy(sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(speNum, d_speNum, twosize, hipMemcpyDeviceToHost);
  hipMemcpy(B, d_a, size, hipMemcpyDeviceToHost);
  hipFree(d_a);hipFree(d_b);hipFree(d_partSum);hipFree(d_ppartSum);hipFree(d_sum);hipFree(d_speNum);


  cout.precision(8);

  // cout<<"time: "<<endbbb<<"   "<<startaaa<<"   "<<CLOCKS_PER_SEC<<endl;
  // cout<<"time: "<<(endTime-startTime)/CLOCKS_PER_SEC<<endl;
  cout<<"time: "<<time<<endl;
  cout<<"Sum: "<<sum[0]<<endl;
  cout<<"A[n/2][n/2]: "<<speNum[0]<<"    "<<A[count/2+N/2]<<"    "<<B[count/2+N/2]<<endl;
  cout<<"A[17][31]: "<<speNum[1]<<"    "<<A[17*N+31]<<"    "<<B[17*N+31]<<endl;
  cout<<"A[999][999]: "<<A[999*N+999]<<"    "<<B[999*N+999]<<endl;
  cout<<"A[999][500]: "<<A[999*N+500]<<"    "<<B[999*N+500]<<endl;
  cout<<"A[500][999]: "<<A[500*N+999]<<"    "<<B[500*N+999]<<endl;
  cout<<"A[500][0]: "<<A[500*N]<<"    "<<B[500*N]<<endl;
  cout<<"A[501][0]: "<<A[501*N]<<"    "<<B[501*N]<<endl;

  return 0;
}
