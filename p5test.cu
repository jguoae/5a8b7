#include "hip/hip_runtime.h"
#include <float.h>
#include <cstdlib>
#include <iostream>
#include <math.h>
#include <ctime>
#include <algorithm>
#include <vector>
using namespace std;

#define N 1000
#define count N*N
#define threadsPerBlock 1000
#define numberBlocks N*N/threadsPerBlock

__device__ int partition(double* input, int start, int end)
{
    double pivot = input[end];

    while(start < end){
        while(input[start] < pivot)
            start++;
        while (input[end] > pivot)
            end--;
        if (input[start] == input[end])
            start++;
        else if(start < end){
            int tmp = input[start];
            input[start] = input[end];
            input[end] = tmp;
        }
    }
    return end;
}

__device__ double quickSelect(double* input, int p, int r, int k)
{
    if(p == r){
      return input[p];
    }
    int j = partition(input, p, r);
    int length = j - p + 1;
    if (length == k){
      return input[j];
    }
    else if( k < length ){
      return quickSelect(input, p, j - 1, k);
    }
    else{
      return quickSelect(input, j + 1, r, k - length);
    }
}

__global__ void median (double *a, double *b) {
  int number = blockIdx.x*blockDim.x + threadIdx.x;

  if((number > N-1) && (number/N != 0) && (number/N != N-1) && (number < N*N-N)){
    double tempCompare[5];
    tempCompare[0] = a[number];
    tempCompare[1] = a[number-1];
    tempCompare[2] = a[number+1];
    tempCompare[3] = a[number-N];
    tempCompare[4] = a[number+N];
    b[number] = quickSelect(tempCompare,0,4,2);
  }
  __syncthreads();
}

__global__ void move (double *b, double *a) {
  int number = blockIdx.x*blockDim.x + threadIdx.x;
  a[number] = b[number];
}

__global__ void reduction (double *in, double *out) {
  __shared__ double temp[threadsPerBlock];
  int id = threadIdx.x;
  temp[id] = in[blockIdx.x*blockDim.x + id];
  if(id<500 && id>11){ temp[id] += temp[id+500]; __syncthreads();}
  if(id<256){ temp[id] += temp[id+256]; __syncthreads();}
  if(id<128){ temp[id] += temp[id+128]; __syncthreads();}
  if(id<64){ temp[id] += temp[id+64]; __syncthreads();}
  if(id<32){ temp[id] += temp[id+32]; __syncthreads();}
  if(id<16){ temp[id] += temp[id+16]; __syncthreads();}
  if(id<8){ temp[id] += temp[id+8]; __syncthreads();}
  if(id<4){ temp[id] += temp[id+4]; __syncthreads();}
  if(id<2){ temp[id] += temp[id+2]; __syncthreads();}
  if(id<1){ temp[id] += temp[id+1]; __syncthreads();}
  if(id==0){out[blockIdx.x] = temp[0];}
}

__global__ void sumGen (double *in, double *out) {
  for(int i=0;i<(N/threadsPerBlock)*(N/threadsPerBlock);i++){
    out[0]+=in[i];
  }
}

__global__ void assign (double *a, double *mid, double *spe) {
  mid[0] = a[count/2+N/2];
  spe[0] = a[17*N+31];
}

int main(){

  double A[count];
  double sum, speNum, midNum;
  double *d_a, *d_b, *d_partSum, *d_ppartSum, *d_sum, *d_speNum, *d_midNum;
  int size = N*N*sizeof(double);

  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      A[i*N+j] = sin(i*i+j)*sin(i*i+j)+cos(i-j);
    }
  }

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_partSum, size/threadsPerBlock);
  hipMalloc((void **)&d_ppartSum, size/threadsPerBlock*threadsPerBlock);
  hipMalloc((void **)&d_sum, sizeof(double));
  hipMemcpy(*d_a, *A, size, hipMemcpyHostToDevice);
  double startTime = clock();

  // for(int i=0;i<10;i++){
  //     median<<<numberBlocks,threadsPerBlock>>>(d_a,d_b);
  //     move<<<numberBlocks,threadsPerBlock>>>(d_b,d_a);
  // }
  reduction<<<count/threadsPerBlock, threadsPerBlock>>>(d_a,d_partSum);
  reduction<<<(count/threadsPerBlock*threadsPerBlock),(count/threadsPerBlock)>>>(d_partSum,d_ppartSum);
  sumGen<<<1,1>>>(d_ppartSum,d_sum);
  assign<<<1,1>>>(d_a, d_speNum, d_midNum);
  hipDeviceSynchronize();

  double endTime = clock();
  hipMemcpy(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&speNum, d_speNum, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&midNum, d_midNum, sizeof(double), hipMemcpyDeviceToHost);
  hipFree(d_a);hipFree(d_b);hipFree(d_partSum);hipFree(d_ppartSum);hipFree(d_sum);hipFree(d_speNum);hipFree(d_midNum);

  cout<<"time: "<<endTime-startTime<<endl;
  cout<<"Sum: "<<sum<<endl;
  cout<<"A[n/2][n/2]: "<<midNum<<endl;
  cout<<"A[17][31]: "<<speNum<<endl;

  return 0;
}
