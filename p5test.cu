#include "hip/hip_runtime.h"
#include <float.h>
#include <cstdlib>
#include <iostream>
#include <math.h>
#include <ctime>
#include <algorithm>
#include <vector>
using namespace std;

#define N = 1000;
#define count = N^2;
#define threadsPerBlock = 1000;
#define numberBlocks = N^2/threadsPerBlock;

__global__ void median (double *a, double *b) {
  int number = blockIdx.x*blockDim.x + threadIdx.x;

  if(number > N-1 && number/N != 0 && number/N != N-1 && number < N^2-N){
    double tempCompare[5];
    // tempCompare[0] = a[number];
    // tempCompare[1] = a[number-1];
    // tempCompare[2] = a[number+1];
    // tempCompare[3] = a[number-N];
    // tempCompare[4] = a[number+N];
    // sort(tempCompare.begin(),tempCompare.end());
    // b[number] = tempCompare[2];
  }
  __syncthreads();
}
/*
__global__ void copy (double *b, double *a) {
  int number = blockIdx.x*blockDim.x + threadIdx.x;
  a[number] = b[number];
}

__global__ void sum (double *in, double *out) {
  __shared__ double temp[threadsPerBlock];
  int id = threadIdx.x;
  temp[id] = in[blockIdx.x*blockDim.x + id];
  if(id<500 && id>11){ temp[id] += temp[id+500]; __syncthreads();}
  if(id<256){ temp[id] += temp[id+256]; __syncthreads();}
  if(id<128){ temp[id] += temp[id+128]; __syncthreads();}
  if(id<64){ temp[id] += temp[id+64]; __syncthreads();}
  if(id<32){ temp[id] += temp[id+32]; __syncthreads();}
  if(id<16){ temp[id] += temp[id+16]; __syncthreads();}
  if(id<8){ temp[id] += temp[id+8]; __syncthreads();}
  if(id<4){ temp[id] += temp[id+4]; __syncthreads();}
  if(id<2){ temp[id] += temp[id+2]; __syncthreads();}
  if(id<1){ temp[id] += temp[id+1]; __syncthreads();}
  if(id==0){out[blockIdx.x] = temp[0];}
}

__global__ void sumGen (double *in, double *out) {
  for(int i=0;i<(N/threadsPerBlock)^2;i++){
    out+=in[i];
  }
}
*/
int main(void){
  /*
  double A[count];
  double B[count];
  double partSum[count/threadsPerBlock];
  double ppartSum[count/threadsPerBlock^2];
  double sum[1];
  double *d_a, *d_b, *d_partSum, *d_ppartSum, *d_sum;
  int size = N*N*sizeof(double);

  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      A[i*N+j] = sin(i^2+j)^2+cos(i-j);
      B[i*N+j] = 0;
    }
  }

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_partSum, size/threadsPerBlock);
  hipMalloc((void **)&d_ppartSum, size/threadsPerBlock^2);
  hipMalloc((void **)&d_sum, sizeof(double));
  hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_partSum, partSum, size/threadsPerBlock, hipMemcpyHostToDevice);
  hipMemcpy(d_ppartSum, ppartSum, size/threadsPerBlock^2, hipMemcpyHostToDevice);
  hipMemcpy(d_sum, sum, sizeof(double), hipMemcpyHostToDevice);
  double startTime = clock();

  for(int i=0;i<10;i++){
      median<<<numberBlocks,threadsPerBlock>>>(d_a,d_b);
      copy<<numberBlocks,threadsPerBlock>>>(d_b,d_a);
  }
  sum<<<N^2/threadsPerBlock, threadsPerBlock>>>(d_a,d_partSum);
  sum<<<N^2/threadsPerBlock^2,N^2/threadsPerBlock>>>(d_partSum,d_ppartSum);
  sumGen<<<1,1>>>(d_ppartSum,d_sum);

  double endTime = clock();
  hipMemcpy(A, d_a, size, hipMemcpyDeviceToHost);
  hipMemcpy(B, d_b, size, hipMemcpyDeviceToHost);
  hipMemcpy(sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
  hipFree(d_a);hipFree(d_b);hipFree(d_partSum);hipFree(d_ppartSum);hipFree(d_sum);

  cout<<"time: "<<endTime-startTime<<endl;
  cout<<"Sum: "<<*sum<<endl;
  cout<<"A[n/2][n/2]: "<<A[N^2/2+N/2]<<endl;
  cout<<"A[17][31]: "<<A[17*N+31]<<endl;
*/
  return 0;
}
