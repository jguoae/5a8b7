#include "hip/hip_runtime.h"
#include <float.h>
#include <cstdlib>
#include <iostream>
#include <math.h>
#include <ctime>
#include <algorithm>
#include <vector>
using namespace std;

#define N 4000
#define count N*N
#define threadsPerBlock 1000
#define numberBlocks N*N/threadsPerBlock

__device__ int partition(double* input, int start, int end)
{
    double pivot = input[end];

    while(start < end){
        while(input[start] < pivot)
            start++;
        while (input[end] > pivot)
            end--;
        if (input[start] == input[end])
            start++;
        else if(start < end){
            double tmp = input[start];
            input[start] = input[end];
            input[end] = tmp;
        }
    }
    return end;
}

__device__ double quickSelect(double* input, int p, int r, int k)
{
    if(p == r){
      return input[p];
    }
    int j = partition(input, p, r);
    int length = j - p + 1;
    if (length == k){
      return input[j];
    }
    else if( k < length ){
      return quickSelect(input, p, j - 1, k);
    }
    else{
      return quickSelect(input, j + 1, r, k - length);
    }
}

__global__ void median (double *a) {//a[n][n]
  int number = blockIdx.x*blockDim.x + threadIdx.x;

  if((number > N-1) && (number/N != 0) && (number/N != N-1) && (number < N*N-N)){
    double tempCompare[5];
    tempCompare[0] = a[number];
    tempCompare[1] = a[number-1];
    tempCompare[2] = a[number+1];
    tempCompare[3] = a[number-N];
    tempCompare[4] = a[number+N];
    a[number] = quickSelect(tempCompare,0,4,2);
    // a[number] = tempCompare[2];
  }
  __syncthreads();
}

// __global__ void move (double *b, double *a) {
//   int number = blockIdx.x*blockDim.x + threadIdx.x;
//   a[number] = b[number];
// }

__global__ void reduction (double *in, double *out) {
  __shared__ double temp[threadsPerBlock];
  int id = threadIdx.x;
  temp[id] = in[blockIdx.x*blockDim.x + id];
  if(id<500 && id>11){ temp[id] += temp[id+500]; __syncthreads();}
  if(id<256){ temp[id] += temp[id+256]; __syncthreads();}
  if(id<128){ temp[id]i2 += temp[id+128]; __syncthreads();}
  if(id<64){ temp[id] += temp[id+64]; __syncthreads();}
  if(id<32){ temp[id] += temp[id+32]; __syncthreads();}
  if(id<16){ temp[id] += temp[id+16]; __syncthreads();}
  if(id<8){ temp[id] += temp[id+8]; __syncthreads();}
  if(id<4){ temp[id] += temp[id+4]; __syncthreads();}
  if(id<2){ temp[id] += temp[id+2]; __syncthreads();}
  if(id<1){ temp[id] += temp[id+1]; __syncthreads();}
  if(id==0){out[blockIdx.x] = temp[0];}
}

__global__ void sumGen (double *in, double *out) {
  for(int i=0;i<(N/threadsPerBlock)*(N/threadsPerBlock);i++){
    out[0]+=in[i];
  }
}

__global__ void assign (double *a, double *spe) {
  spe[0] = a[count/2+N/2];
  spe[1] = a[17*N+31];
}

int main(){
  double A[count], B[count];
  double sum[1], speNum[2];
  double *d_a, *d_partSum, *d_ppartSum, *d_sum, *d_speNum;
  int size = N*N*sizeof(double);
  int twosize = 2*sizeof(double);

  sum[0]=0;

  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      A[i*N+j] = sin(i*i+j)*sin(i*i+j)+cos(i-j);
      B[i*N+j] = 0;
    }
  }
  hipMalloc((void **)&d_a, size);
  // hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_partSum, size/threadsPerBlock);
  hipMalloc((void **)&d_ppartSum, size/threadsPerBlock/threadsPerBlock);
  hipMalloc((void **)&d_sum, sizeof(double));
  hipMalloc((void **)&d_speNum,twosize);
  hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
  double startTime = clock();

  for(int i=0;i<10;i++){
      median<<<numberBlocks,threadsPerBlock>>>(d_a);
      //move<<<numberBlocks,threadsPerBlock>>>(d_b,d_a);
  }
  reduction<<<count/threadsPerBlock, threadsPerBlock>>>(d_a,d_partSum);
  reduction<<<(count/threai2dsPerBlock/threadsPerBlock),threadsPerBlock>>>(d_partSum,d_ppartSum);
  sumGen<<<1,1>>>(d_ppartSum,d_sum);
  assign<<<1,1>>>(d_a, d_speNum);
  // hipDeviceSynchronize();

  double endTime = clock();
  hipMemcpy(sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(speNum, d_speNum, twosize, hipMemcpyDeviceToHost);
  hipMemcpy(B, d_a, size, hipMemcpyDeviceToHost);
  hipFree(d_a);hipFree(d_partSum);hipFree(d_ppartSum);hipFree(d_sum);hipFree(d_speNum);

  cout<<"time: "<<endTime-startTime<<endl;
  cout<<"Sum: "<<sum[0]<<endl;
  cout<<"A[n/2][n/2]: "<<speNum[0]<<"    "<<A[count/2+N/2]<<"    "<<B[count/2+N/2]<<endl;
  cout<<"A[17][31]: "<<speNum[1]<<"    "<<A[17*N+31]<<"    "<<B[17*N+31]<<endl;

  return 0;
}
