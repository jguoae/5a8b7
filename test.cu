
#include <hip/hip_runtime.h>
#include <math.h>
#include <cstdlib>
#include <iostream>
using namespace std;


#define N 512

__global__ void add(int *a, int *b, int *c) {

c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];

}


int main(void) {

int a[N], b[N], c[N]; // host copies of a, b, c
int *d_a, *d_b, *d_c; // device copies of a, b, c
int size = N * sizeof(int);
for(int i=0;i<N;i++){
  a[i]=1;
  b[i]=1;
}

// Alloc space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);
// Alloc space for host copies of a, b, c and setup input values


hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
// Launch add() kernel on GPU with N blocks
add<<<N,1>>>(d_a, d_b, d_c);
// Copy result back to host
hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
// Cleanup
hipFree(d_a); hipFree(d_b); hipFree(d_c);
cout<<c[0]<<endl;
return 0;

}
